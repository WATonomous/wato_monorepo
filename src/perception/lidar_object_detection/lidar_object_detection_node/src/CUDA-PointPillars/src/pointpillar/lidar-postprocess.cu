#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "lidar-postprocess.hpp"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include <algorithm>
#include <math.h>

#include "common/check.hpp"
#include "common/launch.cuh"
#include "common/tensorrt.hpp"

namespace pointpillar {
namespace lidar {

const int NMS_THREADS_PER_BLOCK = sizeof(uint64_t) * 8;
const int DET_CHANNEL = 9;

typedef struct {
  float val[DET_CHANNEL];
} combined_float;

#define DIVUP(x, y) (x + y - 1) / y

__device__ float sigmoid(const float x) { return 1.0f / (1.0f + expf(-x)); }

__global__ void postprocess_kernal(const float *cls_input,
                                        float *box_input,
                                        const float *dir_input,
                                        float *anchors,
                                        float *anchor_bottom_heights,
                                        float *bndbox_output,
                                        float *score_output,
                                        int *object_counter,
                                        const float min_x_range,
                                        const float max_x_range,
                                        const float min_y_range,
                                        const float max_y_range,
                                        const int feature_x_size,
                                        const int feature_y_size,
                                        const int num_anchors,
                                        const int num_classes,
                                        const int num_box_values,
                                        const float score_thresh,
                                        const float dir_offset)
{
  int loc_index = blockIdx.x;
  int ith_anchor = threadIdx.x;
  if (ith_anchor >= num_anchors)
  {
      return;
  }
  int col = loc_index % feature_x_size;
  int row = loc_index / feature_x_size;
  float x_offset = min_x_range + col * (max_x_range - min_x_range) / (feature_x_size - 1);
  float y_offset = min_y_range + row * (max_y_range - min_y_range) / (feature_y_size - 1);
  int cls_offset = loc_index * num_anchors * num_classes + ith_anchor * num_classes;

  const float *scores = cls_input + cls_offset;
  float max_score = sigmoid(scores[0]);
  int cls_id = 0;
  for (int i = 1; i < num_classes; i++) {
    float cls_score = sigmoid(scores[i]);
    if (cls_score > max_score) {
      max_score = cls_score;
      cls_id = i;
    }
  }

  if (max_score >= score_thresh)
  {
    int box_offset = loc_index * num_anchors * num_box_values + ith_anchor * num_box_values;
    int dir_cls_offset = loc_index * num_anchors * 2 + ith_anchor * 2;
    float *anchor_ptr = anchors + ith_anchor * 4;
    float z_offset = anchor_ptr[2] / 2 + anchor_bottom_heights[ith_anchor / 2];
    float anchor[7] = {x_offset, y_offset, z_offset, anchor_ptr[0], anchor_ptr[1], anchor_ptr[2], anchor_ptr[3]};
    float *box_encodings = box_input + box_offset;

    float xa = anchor[0];
    float ya = anchor[1];
    float za = anchor[2];
    float dxa = anchor[3];
    float dya = anchor[4];
    float dza = anchor[5];
    float ra = anchor[6];
    float diagonal = sqrtf(dxa * dxa + dya * dya);
    box_encodings[0] = box_encodings[0] * diagonal + xa;
    box_encodings[1] = box_encodings[1] * diagonal + ya;
    box_encodings[2] = box_encodings[2] * dza + za;
    box_encodings[3] = expf(box_encodings[3]) * dxa;
    box_encodings[4] = expf(box_encodings[4]) * dya;
    box_encodings[5] = expf(box_encodings[5]) * dza;
    box_encodings[6] = box_encodings[6] + ra;

    float yaw;
    int dir_label = dir_input[dir_cls_offset] > dir_input[dir_cls_offset + 1] ? 0 : 1;
    float period = 2 * M_PI / 2;
    float val = box_input[box_offset + 6] - dir_offset;
    float dir_rot = val - floor(val / (period + 1e-8) + 0.f) * period;
    yaw = dir_rot + dir_offset + period * dir_label;

    int resCount = (int)atomicAdd(object_counter, 1);
    float *data = bndbox_output + resCount * 9;
    data[0] = box_input[box_offset];
    data[1] = box_input[box_offset + 1];
    data[2] = box_input[box_offset + 2];
    data[3] = box_input[box_offset + 3];
    data[4] = box_input[box_offset + 4];
    data[5] = box_input[box_offset + 5];
    data[6] = yaw;
    *(int *)&data[7] = cls_id;
    data[8] = max_score;
    score_output[resCount] = max_score;
  }
}

hipError_t postprocess_launch(const float *cls_input,
                      float *box_input,
                      const float *dir_input,
                      float *anchors,
                      float *anchor_bottom_heights,
                      float *bndbox_output,
                      float *score_output,
                      int *object_counter,
                      const float min_x_range,
                      const float max_x_range,
                      const float min_y_range,
                      const float max_y_range,
                      const int feature_x_size,
                      const int feature_y_size,
                      const int num_anchors,
                      const int num_classes,
                      const int num_box_values,
                      const float score_thresh,
                      const float dir_offset,
                      hipStream_t stream)
{
  int bev_size = feature_x_size * feature_y_size;
  dim3 threads (num_anchors);
  dim3 blocks (bev_size);

  postprocess_kernal<<<blocks, threads, 0, stream>>>
                (cls_input,
                 box_input,
                 dir_input,
                 anchors,
                 anchor_bottom_heights,
                 bndbox_output,
                 score_output,
                 object_counter,
                 min_x_range,
                 max_x_range,
                 min_y_range,
                 max_y_range,
                 feature_x_size,
                 feature_y_size,
                 num_anchors,
                 num_classes,
                 num_box_values,
                 score_thresh,
                 dir_offset);
  return hipGetLastError();
}

__device__ inline float cross(const float2 p1, const float2 p2, const float2 p0) {
    return (p1.x - p0.x) * (p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
}

__device__ inline int check_box2d(float const *const box, const float2 p) {
    const float MARGIN = 1e-2;
    float center_x = box[0];
    float center_y = box[1];
    float angle_cos = cos(-box[6]);
    float angle_sin = sin(-box[6]);
    float rot_x = (p.x - center_x) * angle_cos + (p.y - center_y) * (-angle_sin);
    float rot_y = (p.x - center_x) * angle_sin + (p.y - center_y) * angle_cos;

    return (fabs(rot_x) < box[3] / 2 + MARGIN && fabs(rot_y) < box[4] / 2 + MARGIN);
}

__device__ inline bool intersection(const float2 p1, const float2 p0, const float2 q1, const float2 q0, float2 &ans) {

    if (( fmin(p0.x, p1.x) <= fmax(q0.x, q1.x) &&
          fmin(q0.x, q1.x) <= fmax(p0.x, p1.x) &&
          fmin(p0.y, p1.y) <= fmax(q0.y, q1.y) &&
          fmin(q0.y, q1.y) <= fmax(p0.y, p1.y) ) == 0)
        return false;


    float s1 = cross(q0, p1, p0);
    float s2 = cross(p1, q1, p0);
    float s3 = cross(p0, q1, q0);
    float s4 = cross(q1, p1, q0);

    if (!(s1 * s2 > 0 && s3 * s4 > 0))
        return false;

    float s5 = cross(q1, p1, p0);
    if (fabs(s5 - s1) > 1e-8) {
        ans.x = (s5 * q0.x - s1 * q1.x) / (s5 - s1);
        ans.y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);

    } else {
        float a0 = p0.y - p1.y, b0 = p1.x - p0.x, c0 = p0.x * p1.y - p1.x * p0.y;
        float a1 = q0.y - q1.y, b1 = q1.x - q0.x, c1 = q0.x * q1.y - q1.x * q0.y;
        float D = a0 * b1 - a1 * b0;

        ans.x = (b0 * c1 - b1 * c0) / D;
        ans.y = (a1 * c0 - a0 * c1) / D;
    }

    return true;
}

__device__ inline void rotate_around_center(const float2 &center, const float angle_cos, const float angle_sin, float2 &p) {
    float new_x = (p.x - center.x) * angle_cos + (p.y - center.y) * (-angle_sin) + center.x;
    float new_y = (p.x - center.x) * angle_sin + (p.y - center.y) * angle_cos + center.y;
    p = float2 {new_x, new_y};
    return;
}

__device__ inline bool devIoU(float const *const box_a, float const *const box_b, const float nms_thresh) {
    float a_angle = box_a[6], b_angle = box_b[6];
    float a_dx_half = box_a[3] / 2, b_dx_half = box_b[3] / 2, a_dy_half = box_a[4] / 2, b_dy_half = box_b[4] / 2;
    float a_x1 = box_a[0] - a_dx_half, a_y1 = box_a[1] - a_dy_half;
    float a_x2 = box_a[0] + a_dx_half, a_y2 = box_a[1] + a_dy_half;
    float b_x1 = box_b[0] - b_dx_half, b_y1 = box_b[1] - b_dy_half;
    float b_x2 = box_b[0] + b_dx_half, b_y2 = box_b[1] + b_dy_half;
    float2 box_a_corners[5];
    float2 box_b_corners[5];

    float2 center_a = float2 {box_a[0], box_a[1]};
    float2 center_b = float2 {box_b[0], box_b[1]};

    float2 cross_points[16];
    float2 poly_center =  {0, 0};
    int cnt = 0;
    bool flag = false;

    box_a_corners[0] = float2 {a_x1, a_y1};
    box_a_corners[1] = float2 {a_x2, a_y1};
    box_a_corners[2] = float2 {a_x2, a_y2};
    box_a_corners[3] = float2 {a_x1, a_y2};

    box_b_corners[0] = float2 {b_x1, b_y1};
    box_b_corners[1] = float2 {b_x2, b_y1};
    box_b_corners[2] = float2 {b_x2, b_y2};
    box_b_corners[3] = float2 {b_x1, b_y2};

    float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
    float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

    for (int k = 0; k < 4; k++) {
        rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
        rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
    }

    box_a_corners[4] = box_a_corners[0];
    box_b_corners[4] = box_b_corners[0];

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            flag = intersection(box_a_corners[i + 1], box_a_corners[i],
                                box_b_corners[j + 1], box_b_corners[j],
                                cross_points[cnt]);
            if (flag) {
                poly_center = {poly_center.x + cross_points[cnt].x, poly_center.y + cross_points[cnt].y};
                cnt++;
            }
        }
    }

    for (int k = 0; k < 4; k++) {
        if (check_box2d(box_a, box_b_corners[k])) {
            poly_center = {poly_center.x + box_b_corners[k].x, poly_center.y + box_b_corners[k].y};
            cross_points[cnt] = box_b_corners[k];
            cnt++;
        }
        if (check_box2d(box_b, box_a_corners[k])) {
            poly_center = {poly_center.x + box_a_corners[k].x, poly_center.y + box_a_corners[k].y};
            cross_points[cnt] = box_a_corners[k];
            cnt++;
        }
    }

    poly_center.x /= cnt;
    poly_center.y /= cnt;

    float2 temp;
    for (int j = 0; j < cnt - 1; j++) {
        for (int i = 0; i < cnt - j - 1; i++) {
            if (atan2(cross_points[i].y - poly_center.y, cross_points[i].x - poly_center.x) >
                atan2(cross_points[i+1].y - poly_center.y, cross_points[i+1].x - poly_center.x)
                ) {
                temp = cross_points[i];
                cross_points[i] = cross_points[i + 1];
                cross_points[i + 1] = temp;
            }
        }
    }

    float area = 0;
    for (int k = 0; k < cnt - 1; k++) {
        float2 a = {cross_points[k].x - cross_points[0].x,
                    cross_points[k].y - cross_points[0].y};
        float2 b = {cross_points[k + 1].x - cross_points[0].x,
                    cross_points[k + 1].y - cross_points[0].y};
        area += (a.x * b.y - a.y * b.x);
    }

    float s_overlap = fabs(area) / 2.0;;
    float sa = box_a[3] * box_a[4];
    float sb = box_b[3] * box_b[4];
    float iou = s_overlap / fmaxf(sa + sb - s_overlap, 1e-8);

    return iou >= nms_thresh;
}

__global__ void nms_cuda(const int n_boxes, const float iou_threshold, const float *dev_boxes, uint64_t *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;
  const int tid = threadIdx.x;

  if (row_start > col_start) return;

  const int row_size = fminf(n_boxes - row_start * NMS_THREADS_PER_BLOCK, NMS_THREADS_PER_BLOCK);
  const int col_size = fminf(n_boxes - col_start * NMS_THREADS_PER_BLOCK, NMS_THREADS_PER_BLOCK);

  __shared__ float block_boxes[NMS_THREADS_PER_BLOCK * 7];

  if (tid < col_size) {
    int idx = NMS_THREADS_PER_BLOCK * col_start + tid;
    block_boxes[tid * 7 + 0] = dev_boxes[idx * DET_CHANNEL + 0];
    block_boxes[tid * 7 + 1] = dev_boxes[idx * DET_CHANNEL + 1];
    block_boxes[tid * 7 + 2] = dev_boxes[idx * DET_CHANNEL + 2];
    block_boxes[tid * 7 + 3] = dev_boxes[idx * DET_CHANNEL + 3];
    block_boxes[tid * 7 + 4] = dev_boxes[idx * DET_CHANNEL + 4];
    block_boxes[tid * 7 + 5] = dev_boxes[idx * DET_CHANNEL + 5];
    block_boxes[tid * 7 + 6] = dev_boxes[idx * DET_CHANNEL + 6];
  }
  __syncthreads();

  if (tid < row_size) {
    const int cur_box_idx = NMS_THREADS_PER_BLOCK * row_start + tid;
    const float *cur_box = dev_boxes + cur_box_idx * DET_CHANNEL;
    int i = 0;
    uint64_t t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = tid + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 7, iou_threshold)) {
        t |= 1ULL << i;
      }
    }
    dev_mask[cur_box_idx * gridDim.y + col_start] = t;
  }
}

hipError_t nms_launch(unsigned int boxes_num,
               float *boxes,
               float nms_thresh,
               uint64_t* mask,
               hipStream_t stream)
{
    int col_blocks = DIVUP(boxes_num, NMS_THREADS_PER_BLOCK);

    dim3 blocks(col_blocks, col_blocks);
    dim3 threads(NMS_THREADS_PER_BLOCK);

    nms_cuda<<<blocks, threads, 0, stream>>>(boxes_num, nms_thresh, boxes, mask);

    return hipGetLastError();
}

class PostProcessImplement : public PostProcess {
public:
    virtual ~PostProcessImplement() {
        if (bndbox_) checkRuntime(hipFree(bndbox_));
        if (h_bndbox_) checkRuntime(hipHostFree(h_bndbox_));
        if (score_) checkRuntime(hipFree(score_));

        if (anchors_) checkRuntime(hipFree(anchors_));
        if (anchor_bottom_heights_) checkRuntime(hipFree(anchor_bottom_heights_));
        if (object_counter_) checkRuntime(hipFree(object_counter_));

        if (h_mask_) checkRuntime(hipHostFree(h_mask_));
    }

    virtual bool init(const PostProcessParameter& param) {
        param_ = param;

        det_num_ = param_.feature_size.x * param_.feature_size.y * param_.num_anchors;
        checkRuntime(hipMalloc((void **)&bndbox_, det_num_ * 9 * sizeof(float)));
        checkRuntime(hipHostMalloc((void **)&h_bndbox_, det_num_ * 9 * sizeof(float)));
        checkRuntime(hipMalloc((void **)&score_, det_num_ * sizeof(float)));

        checkRuntime(hipMalloc((void **)&anchors_, param_.num_anchors * param_.len_per_anchor * sizeof(float)));
        checkRuntime(hipMalloc((void **)&anchor_bottom_heights_, param_.num_classes * sizeof(float)));
        checkRuntime(hipMalloc((void **)&object_counter_, sizeof(int)));

        checkRuntime(hipMemcpy(anchors_, param_.anchors, param_.num_anchors * param_.len_per_anchor * sizeof(float), hipMemcpyDefault));
        checkRuntime(hipMemcpy(anchor_bottom_heights_, &param_.anchor_bottom_heights, param_.num_classes * sizeof(float), hipMemcpyDefault));

        h_mask_size_ = det_num_ * DIVUP(det_num_, NMS_THREADS_PER_BLOCK) * sizeof(uint64_t);
        checkRuntime(hipHostMalloc((void **)&h_mask_, h_mask_size_));

        int res_blocks = DIVUP(det_num_, NMS_THREADS_PER_BLOCK);
        remv_ = std::vector<uint64_t>(res_blocks, 0);
        bndbox_after_nms_.resize(det_num_);

        return true;
    }

    virtual void forward(const float* cls, const float* box, const float* dir, void* stream) override {
        hipStream_t _stream = static_cast<hipStream_t>(stream);

        checkRuntime(hipMemsetAsync(object_counter_, 0, sizeof(int), _stream));
        checkRuntime(hipMemsetAsync(h_mask_, 0, h_mask_size_, _stream));

        checkRuntime(postprocess_launch((float *)cls,
                                        (float *)box,
                                        (float *)dir,
                                        anchors_,
                                        anchor_bottom_heights_,
                                        bndbox_,
                                        score_,
                                        object_counter_,
                                        param_.min_range.x,
                                        param_.max_range.x,
                                        param_.min_range.y,
                                        param_.max_range.y,
                                        param_.feature_size.x,
                                        param_.feature_size.y,
                                        param_.num_anchors,
                                        param_.num_classes,
                                        param_.num_box_values,
                                        param_.score_thresh,
                                        param_.dir_offset,
                                        _stream
                                        ));
        checkRuntime(hipMemcpyAsync(&bndbox_num_, object_counter_, sizeof(int), hipMemcpyDeviceToHost, _stream));
        checkRuntime(hipStreamSynchronize(_stream));

        thrust::device_ptr<combined_float> thr_bndbox_((combined_float *)bndbox_);
        thrust::stable_sort_by_key(thrust::cuda::par.on(_stream), score_, score_ + bndbox_num_, thr_bndbox_, thrust::greater<float>());
        checkRuntime(nms_launch(bndbox_num_, bndbox_, param_.nms_thresh, h_mask_, _stream));

        checkRuntime(hipMemcpyAsync(h_bndbox_, bndbox_, bndbox_num_ * 9 * sizeof(float), hipMemcpyDeviceToHost, _stream));
        checkRuntime(hipStreamSynchronize(_stream));

        int col_blocks = DIVUP(bndbox_num_, NMS_THREADS_PER_BLOCK);
        memset(remv_.data(), 0, col_blocks * sizeof(uint64_t));
        bndbox_num_after_nms_ = 0;

        for (unsigned int i_nms = 0; i_nms < bndbox_num_; i_nms++) {
            unsigned int nblock = i_nms / NMS_THREADS_PER_BLOCK;
            unsigned int inblock = i_nms % NMS_THREADS_PER_BLOCK;

            if (!(remv_[nblock] & (1ULL << inblock))) {
                bndbox_after_nms_[bndbox_num_after_nms_] = *(BoundingBox*)(&h_bndbox_[i_nms * 9]);
                bndbox_num_after_nms_++;
                uint64_t* p = h_mask_ + i_nms * col_blocks;
                for (int j_nms = nblock; j_nms < col_blocks; j_nms++) {
                    remv_[j_nms] |= p[j_nms];
                }
            }
        }
    }

    virtual std::vector<BoundingBox> bndBoxVec() override {
        return std::vector<BoundingBox>(bndbox_after_nms_.begin(), bndbox_after_nms_.begin() + bndbox_num_after_nms_);
    }

private:
    PostProcessParameter param_;
    float *anchors_;
    float *anchor_bottom_heights_;
    int *object_counter_;

    float *bndbox_ = nullptr;
    float *h_bndbox_ = nullptr;
    float *score_ = nullptr;
    unsigned int det_num_ = 0;

    uint64_t* h_mask_ = nullptr;
    unsigned int h_mask_size_ = 0;
    std::vector<uint64_t> remv_;

    unsigned int bndbox_num_ = 0;
    std::vector<BoundingBox> bndbox_after_nms_;
    unsigned int bndbox_num_after_nms_ = 0;
};

std::shared_ptr<PostProcess> create_postprocess(const PostProcessParameter& param) {
  std::shared_ptr<PostProcessImplement> instance(new PostProcessImplement());
  if (!instance->init(param)) {
    instance.reset();
  }
  return instance;
}

};  // namespace lidar
};  // namespace pointpillar