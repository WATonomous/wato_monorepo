#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_fp16.h>
#include "lidar-voxelization.hpp"

#include "common/check.hpp"
#include "common/launch.cuh"


namespace pointpillar {
namespace lidar {

const int POINTS_PER_VOXEL = 32;
const int WARP_SIZE = 32;
const int WARPS_PER_BLOCK = 4;
const int FEATURES_SIZE = 10;

static __global__ void generateVoxels_random_kernel(const float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, float *voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  float4 point = ((float4*)points)[point_idx];

  if(point.x<min_x_range||point.x>=max_x_range
    || point.y<min_y_range||point.y>=max_y_range
    || point.z<min_z_range||point.z>=max_z_range) return;

  int voxel_idx = floorf((point.x - min_x_range)/pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range)/pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size
                            + voxel_idx;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if(point_id >= POINTS_PER_VOXEL) return;
  float *address = voxels + (voxel_index*POINTS_PER_VOXEL + point_id)*4;
  atomicExch(address+0, point.x);
  atomicExch(address+1, point.y);
  atomicExch(address+2, point.z);
  atomicExch(address+3, point.w);
}

hipError_t generateVoxels_random_launch(const float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, float *voxels,
        hipStream_t stream)
{
  dim3 blocks((points_size+256-1)/256);
  dim3 threads(256);
  generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>
    (points, points_size,
        min_x_range, max_x_range,
        min_y_range, max_y_range,
        min_z_range, max_z_range,
        pillar_x_size, pillar_y_size, pillar_z_size,
        grid_y_size, grid_x_size,
        mask, voxels);
  hipError_t err = hipGetLastError();
  return err;
}

static __global__ void generateBaseFeatures_kernel(unsigned int *mask, float *voxels,
        int grid_y_size, int grid_x_size,
        unsigned int *pillar_num,
        float *voxel_features,
        unsigned int *voxel_num,
        unsigned int *voxel_idxs)
{
  unsigned int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if(voxel_idx >= grid_x_size ||voxel_idy >= grid_y_size) return;

  unsigned int voxel_index = voxel_idy * grid_x_size
                           + voxel_idx;
  unsigned int count = mask[voxel_index];
  if( !(count>0) ) return;
  count = count<POINTS_PER_VOXEL?count:POINTS_PER_VOXEL;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);

  voxel_num[current_pillarId] = count;

  uint4 idx = {0, 0, voxel_idy, voxel_idx};
  ((uint4*)voxel_idxs)[current_pillarId] = idx;

  for (int i=0; i<count; i++){
    int inIndex = voxel_index*POINTS_PER_VOXEL + i;
    int outIndex = current_pillarId*POINTS_PER_VOXEL + i;
    ((float4*)voxel_features)[outIndex] = ((float4*)voxels)[inIndex];
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t generateBaseFeatures_launch(unsigned int *mask, float *voxels,
        int grid_y_size, int grid_x_size,
        unsigned int *pillar_num,
        float *voxel_features,
        unsigned int *voxel_num,
        unsigned int *voxel_idxs,
        hipStream_t stream)
{
  dim3 threads = {32,32};
  dim3 blocks = {(grid_x_size + threads.x -1)/threads.x,
                 (grid_y_size + threads.y -1)/threads.y};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream>>>
      (mask, voxels, grid_y_size, grid_x_size,
       pillar_num,
       voxel_features,
       voxel_num,
       voxel_idxs);
  hipError_t err = hipGetLastError();
  return err;
}

// 4 channels -> 10 channels
static __global__ void generateFeatures_kernel(float* voxel_features,
    unsigned int* voxel_num, unsigned int* voxel_idxs, unsigned int *params,
    float voxel_x, float voxel_y, float voxel_z,
    float range_min_x, float range_min_y, float range_min_z,
    half* features)
{
    int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x/WARP_SIZE;
    int point_idx = threadIdx.x % WARP_SIZE;

    int pillar_idx_inBlock = threadIdx.x/WARP_SIZE;
    unsigned int num_pillars = params[0];

    if (pillar_idx >= num_pillars) return;

    __shared__ float4 pillarSM[WARPS_PER_BLOCK][WARP_SIZE];
    __shared__ float4 pillarSumSM[WARPS_PER_BLOCK];
    __shared__ uint4 idxsSM[WARPS_PER_BLOCK];
    __shared__ int pointsNumSM[WARPS_PER_BLOCK];
    __shared__ half pillarOutSM[WARPS_PER_BLOCK][WARP_SIZE][FEATURES_SIZE];

    if (threadIdx.x < WARPS_PER_BLOCK) {
      pointsNumSM[threadIdx.x] = voxel_num[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
      idxsSM[threadIdx.x] = ((uint4*)voxel_idxs)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
      pillarSumSM[threadIdx.x] = {0,0,0,0};
    }

    pillarSM[pillar_idx_inBlock][point_idx] = ((float4*)voxel_features)[pillar_idx*WARP_SIZE + point_idx];
    __syncthreads();

    //calculate sm in a pillar
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x),  pillarSM[pillar_idx_inBlock][point_idx].x);
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y),  pillarSM[pillar_idx_inBlock][point_idx].y);
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z),  pillarSM[pillar_idx_inBlock][point_idx].z);
    }
    __syncthreads();

    //feature-mean
    float4 mean;
    float validPoints = pointsNumSM[pillar_idx_inBlock];
    mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
    mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
    mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

    mean.x  = pillarSM[pillar_idx_inBlock][point_idx].x - mean.x;
    mean.y  = pillarSM[pillar_idx_inBlock][point_idx].y - mean.y;
    mean.z  = pillarSM[pillar_idx_inBlock][point_idx].z - mean.z;


    //calculate offset
    float x_offset = voxel_x / 2 + idxsSM[pillar_idx_inBlock].w * voxel_x + range_min_x;
    float y_offset = voxel_y / 2 + idxsSM[pillar_idx_inBlock].z * voxel_y + range_min_y;
    float z_offset = voxel_z / 2 + idxsSM[pillar_idx_inBlock].y * voxel_z + range_min_z;

    //feature-offset
    float4 center;
    center.x  = pillarSM[pillar_idx_inBlock][point_idx].x - x_offset;
    center.y  = pillarSM[pillar_idx_inBlock][point_idx].y - y_offset;
    center.z  = pillarSM[pillar_idx_inBlock][point_idx].z - z_offset;

    //store output
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
      pillarOutSM[pillar_idx_inBlock][point_idx][0] = __float2half(pillarSM[pillar_idx_inBlock][point_idx].x);
      pillarOutSM[pillar_idx_inBlock][point_idx][1] = __float2half(pillarSM[pillar_idx_inBlock][point_idx].y);
      pillarOutSM[pillar_idx_inBlock][point_idx][2] = __float2half(pillarSM[pillar_idx_inBlock][point_idx].z);
      pillarOutSM[pillar_idx_inBlock][point_idx][3] = __float2half(pillarSM[pillar_idx_inBlock][point_idx].w);

      pillarOutSM[pillar_idx_inBlock][point_idx][4] = __float2half(mean.x);
      pillarOutSM[pillar_idx_inBlock][point_idx][5] = __float2half(mean.y);
      pillarOutSM[pillar_idx_inBlock][point_idx][6] = __float2half(mean.z);

      pillarOutSM[pillar_idx_inBlock][point_idx][7] = __float2half(center.x);
      pillarOutSM[pillar_idx_inBlock][point_idx][8] = __float2half(center.y);
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = __float2half(center.z);

    } else {
      pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;

      pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;

      pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = 0;
    }

    __syncthreads();

    for(int i = 0; i < FEATURES_SIZE; i ++) {
      int outputSMId = pillar_idx_inBlock*WARP_SIZE*FEATURES_SIZE + i* WARP_SIZE + point_idx;
      int outputId = pillar_idx*WARP_SIZE*FEATURES_SIZE + i* WARP_SIZE + point_idx;
      features[outputId] = ((half*)pillarOutSM)[outputSMId];
    }

}

nvtype::Int3 VoxelizationParameter::compute_grid_size(const nvtype::Float3 &max_range, const nvtype::Float3 &min_range,
                                                      const nvtype::Float3 &voxel_size) {
  nvtype::Int3 size;
  size.x = static_cast<int>(std::round((max_range.x - min_range.x) / voxel_size.x));
  size.y = static_cast<int>(std::round((max_range.y - min_range.y) / voxel_size.y));
  size.z = static_cast<int>(std::round((max_range.z - min_range.z) / voxel_size.z));
  return size;
}

hipError_t generateFeatures_launch(float* voxel_features,
    unsigned int * voxel_num,
    unsigned int* voxel_idxs,
    unsigned int *params, unsigned int max_voxels,
    float voxel_x, float voxel_y, float voxel_z,
    float range_min_x, float range_min_y, float range_min_z,
    nvtype::half* features,
    hipStream_t stream)
{
    dim3 blocks((max_voxels+WARPS_PER_BLOCK-1)/WARPS_PER_BLOCK);
    dim3 threads(WARPS_PER_BLOCK*WARP_SIZE);

    generateFeatures_kernel<<<blocks, threads, 0, stream>>>
      (voxel_features,
      voxel_num,
      voxel_idxs,
      params,
      voxel_x, voxel_y, voxel_z,
      range_min_x, range_min_y, range_min_z,
      (half *)features);

    hipError_t err = hipGetLastError();
    return err;
}

class VoxelizationImplement : public Voxelization {
    public:
        virtual ~VoxelizationImplement() {
            if (voxel_features_) checkRuntime(hipFree(voxel_features_));
            if (voxel_num_) checkRuntime(hipFree(voxel_num_));
            if (voxel_idxs_) checkRuntime(hipFree(voxel_idxs_));

            if (features_input_) checkRuntime(hipFree(features_input_));
            if (params_input_) checkRuntime(hipFree(params_input_));

            if (mask_) checkRuntime(hipFree(mask_));
            if (voxels_) checkRuntime(hipFree(voxels_));
            if (voxelsList_) checkRuntime(hipFree(voxelsList_));
        }

    bool init(VoxelizationParameter param) {
        param_ = param;

        mask_size_ = param_.grid_size.z * param_.grid_size.y
                    * param_.grid_size.x * sizeof(unsigned int);
        voxels_size_ = param_.grid_size.z * param_.grid_size.y * param_.grid_size.x
                    * param_.max_points_per_voxel * 4 * sizeof(float);
        voxel_features_size_ = param_.max_voxels * param_.max_points_per_voxel * 4 * sizeof(float);
        voxel_num_size_ = param_.max_voxels * sizeof(unsigned int);
        voxel_idxs_size_ = param_.max_voxels * 4 * sizeof(unsigned int);
        features_input_size_ = param_.max_voxels * param_.max_points_per_voxel * 10 * sizeof(nvtype::half);

        checkRuntime(hipMalloc((void **)&voxel_features_, voxel_features_size_));
        checkRuntime(hipMalloc((void **)&voxel_num_, voxel_num_size_));

        checkRuntime(hipMalloc((void **)&features_input_, features_input_size_));
        checkRuntime(hipMalloc((void **)&voxel_idxs_, voxel_idxs_size_));
        checkRuntime(hipMalloc((void **)&params_input_, sizeof(unsigned int)));

        checkRuntime(hipMalloc((void **)&mask_, mask_size_));
        checkRuntime(hipMalloc((void **)&voxels_, voxels_size_));
        checkRuntime(hipMalloc((void **)&voxelsList_, param_.max_points * sizeof(int)));

        checkRuntime(hipMemset(voxel_features_, 0, voxel_features_size_));
        checkRuntime(hipMemset(voxel_num_, 0, voxel_num_size_));

        checkRuntime(hipMemset(mask_, 0, mask_size_));
        checkRuntime(hipMemset(voxels_, 0, voxels_size_));
        checkRuntime(hipMemset(voxelsList_, 0, param_.max_points * sizeof(int)));

        checkRuntime(hipMemset(features_input_, 0, features_input_size_));
        checkRuntime(hipMemset(voxel_idxs_, 0, voxel_idxs_size_));

        return true;
    }

    // points and voxels must be of half type
    virtual void forward(const float *_points, int num_points, void *stream) override {
        hipStream_t _stream = reinterpret_cast<hipStream_t>(stream);

        checkRuntime(hipMemsetAsync(params_input_, 0, sizeof(unsigned int), _stream));

        checkRuntime(generateVoxels_random_launch(_points, num_points,
                    param_.min_range.x, param_.max_range.x,
                    param_.min_range.y, param_.max_range.y,
                    param_.min_range.z, param_.max_range.z,
                    param_.voxel_size.x, param_.voxel_size.y, param_.voxel_size.z,
                    param_.grid_size.y, param_.grid_size.x,
                    mask_, voxels_, _stream));

        checkRuntime(generateBaseFeatures_launch(mask_, voxels_,
                    param_.grid_size.y, param_.grid_size.x,
                    params_input_,
                    voxel_features_,
                    voxel_num_,
                    voxel_idxs_, _stream));

        checkRuntime(generateFeatures_launch(voxel_features_,
                    voxel_num_,
                    voxel_idxs_,
                    params_input_, param_.max_voxels,
                    param_.voxel_size.x, param_.voxel_size.y, param_.voxel_size.z,
                    param_.min_range.x, param_.min_range.y, param_.min_range.z,
                    features_input_, _stream));
    }

    virtual const nvtype::half *features() override { return features_input_; }

    virtual const unsigned int *coords() override { return voxel_idxs_; }

    virtual const unsigned int *params() override { return params_input_; }

    private:
        VoxelizationParameter param_;
        
        unsigned int *mask_ = nullptr;
        float *voxels_ = nullptr;
        int *voxelsList_ = nullptr;
        float *voxel_features_ = nullptr;
        unsigned int *voxel_num_ = nullptr;

        nvtype::half *features_input_ = nullptr;
        unsigned int *voxel_idxs_ = nullptr;
        unsigned int *params_input_ = nullptr;

        unsigned int mask_size_;
        unsigned int voxels_size_;
        unsigned int voxel_features_size_;
        unsigned int voxel_num_size_;
        unsigned int voxel_idxs_size_;
        unsigned int features_input_size_ = 0;
};

std::shared_ptr<Voxelization> create_voxelization(VoxelizationParameter param) {
  std::shared_ptr<VoxelizationImplement> impl(new VoxelizationImplement());
  if (!impl->init(param)) {
    impl.reset();
  }
  return impl;
}

};  // namespace lidar
};  // namespace pointpillar
